#include "hip/hip_runtime.h"
#include <stdio.h>
#include "gpu.h"
#include "func_cuda.h"

#define ALLOC_SIZE      1000000000UL

extern __global__ void
delay(volatile int *flag, unsigned long long timeout_clocks = 1000000000);

static int checkp2p(int gpu, int peergpu) {
    int access;
    checkCudaError(hipDeviceCanAccessPeer(&access, gpu, peergpu));
    return access;
}

void unidp2p(int size, int src, int dst)
{
    printf("Testing memory copy from GPU %d to GPU %d\n", src, dst);

    int p2p_enable;
    GPU_t gpu, peergpu;

    initGPUResource(&gpu, p2p_src, src);
    initGPUResource(&peergpu, p2p_dst, dst);

    if (p2p_enable = checkp2p(gpu.id, peergpu.id)) {
        checkCudaError(hipSetDevice(gpu.id));
        checkCudaError(hipDeviceEnablePeerAccess(peergpu.id, 0));
        checkCudaError(hipSetDevice(peergpu.id));
        checkCudaError(hipDeviceEnablePeerAccess(gpu.id, 0));
    }
    else {
        fprintf(stderr, "Warning: GPU %d can't directly access GPU %d\n", src, dst);
    }

    volatile int *flag = NULL;
    checkCudaError(hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable));

    // Arrange task to stream
    printf("Arrange GPU tasks\n");

    *flag = 0;
    delay<<<1, 1, 0, gpu.stream>>>(flag);
    checkCudaError(hipEventRecord(gpu.start, gpu.stream));
    for (int i = 0; i < size; i++)
        checkCudaError(hipMemcpyAsync(peergpu.d_odata, gpu.d_idata, ALLOC_SIZE,
                       hipMemcpyDefault, gpu.stream));
    checkCudaError(hipEventRecord(gpu.stop, gpu.stream));

    // Release stream
    *flag = 1;
    checkCudaError(hipStreamSynchronize(gpu.stream));

    // Output result and free resources
    float time_ms, time, bandwidth;

    checkCudaError(hipEventElapsedTime(&time_ms, gpu.start, gpu.stop));
    time = time_ms / (float) 1e3;
    bandwidth = size / time;
    printf("%20s,%15s\n", "Bandwidth (GB/s)", "Latency (s)");
    printf("%20.2f,%15.2f\n", bandwidth, time / size);

    if (p2p_enable) {
        checkCudaError(hipSetDevice(gpu.id));
        checkCudaError(hipDeviceDisablePeerAccess(peergpu.id));

        checkCudaError(hipSetDevice(peergpu.id));
        checkCudaError(hipDeviceDisablePeerAccess(gpu.id));
    }

    freeGPUResource(&gpu, p2p_src);
    freeGPUResource(&peergpu, p2p_dst);

    checkCudaError(hipHostFree((void *) flag));
}

void bidp2p(int size, int src, int dst)
{
    printf("Testing memory copy between GPU %d and GPU %d\n", src, dst);

    int p2p_enable;
    GPU_t gpu, peergpu;

    initGPUResource(&gpu, bid_p2p, src);
    initGPUResource(&peergpu, bid_p2p, dst);

    if (p2p_enable = checkp2p(gpu.id, peergpu.id)) {
        checkCudaError(hipSetDevice(gpu.id));
        checkCudaError(hipDeviceEnablePeerAccess(peergpu.id, 0));
        checkCudaError(hipSetDevice(peergpu.id));
        checkCudaError(hipDeviceEnablePeerAccess(gpu.id, 0));
    }
    else {
        fprintf(stderr, "Warning: GPU: %d can't directly access GPU: %d\n", src, dst);
    }

    volatile int *flag = NULL;
    checkCudaError(hipHostAlloc((void **)&flag, sizeof(*flag), hipHostMallocPortable));

    // Arrange task to each stream
    printf("Arrange GPU tasks\n");

    *flag = 0;
    delay<<<1, 1, 0, gpu.stream>>>(flag);
    checkCudaError(hipEventRecord(gpu.start, gpu.stream));
    checkCudaError(hipStreamWaitEvent(peergpu.stream, gpu.start, 0));
    for (int i = 0; i < size; i++) {
        checkCudaError(hipMemcpyAsync(peergpu.d_odata, gpu.d_idata, ALLOC_SIZE, hipMemcpyDefault, gpu.stream));
        checkCudaError(hipMemcpyAsync(gpu.d_odata, peergpu.d_idata, ALLOC_SIZE, hipMemcpyDefault, peergpu.stream));
    }
    checkCudaError(hipEventRecord(peergpu.stop, peergpu.stream));
    checkCudaError(hipStreamWaitEvent(gpu.stream, peergpu.stop, 0));
    checkCudaError(hipEventRecord(gpu.stop, gpu.stream));

    *flag = 1;
    checkCudaError(hipStreamSynchronize(gpu.stream));
    checkCudaError(hipStreamSynchronize(peergpu.stream));

    // Output result and free resources
    float time_ms, time, bandwidth;

    checkCudaError(hipEventElapsedTime(&time_ms, gpu.start, gpu.stop));
    time = time_ms / (float) 1e3;
    bandwidth = 2 * size / time;
    printf("%20s,%15s\n", "Bandwidth (GB/s)", "Latency (s)");
    printf("%20.2f,%15.2f\n", bandwidth, time / size);

    if (p2p_enable) {
        checkCudaError(hipSetDevice(gpu.id));
        checkCudaError(hipDeviceDisablePeerAccess(peergpu.id));
        checkCudaError(hipSetDevice(peergpu.id));
        checkCudaError(hipDeviceDisablePeerAccess(gpu.id));
    }

    freeGPUResource(&peergpu, p2p_dst);
    freeGPUResource(&gpu, p2p_src);

    checkCudaError(hipHostFree((void *) flag));
}
